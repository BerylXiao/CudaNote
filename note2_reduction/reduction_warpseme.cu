#include <stdio.h>
#include <hip/hip_runtime.h>

#include <time.h>
#include <limits>

// CUDA错误检查宏（保持不变）
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA Error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
            exit(1); \
        } \
    } while (0)

// 假设输入数组：{1,2,3,4,5,6,7,8}
// 使用2个块(block)，每个块64个线程，blockSize=64
template<typename T, int blockSize>
__device__ void WarpSharedMemReduce(volatile T* smem, int tid) {
    T x = smem[tid];
    if (blockSize >= 64) {
        x += smem[tid + 32]; __syncwarp();
        smem[tid] = x; __syncwarp();
    }
    x += smem[tid + 16]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 8]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 4]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 2]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 1]; __syncwarp();
    smem[tid] = x; __syncwarp();
}

template<typename T, int blockSize>
__global__ void reduce_warpseme(T *d_in, T *d_out, int N) {
    __shared__ T smem[blockSize];
    
    int tid = threadIdx.x;
    int gtid = blockIdx.x * (blockSize * 2) + threadIdx.x;
    
    /* 第一步：加载数据到共享内存并完成第一次规约
    以块0为例，假设blockSize=64：
    - 线程0-63加载并规约128个数据
    - 每个线程处理相距64位置的两个数
    - 如tid=0: smem[0] = d_in[0] + d_in[64]
    - 如tid=1: smem[1] = d_in[1] + d_in[65]
    等等
    */
    if (gtid + blockSize < N) {
        smem[tid] = d_in[gtid] + d_in[gtid + blockSize];
    } else {
        smem[tid] = (gtid < N) ? d_in[gtid] : T(0);
    }
    __syncthreads();
    
    /* 第二步：规约计算(非warp部分)
    - 从blockSize/2开始，一直规约到32个元素
    - 例如：从64个数规约到32个数
    */
    for(int s = blockSize/2; s > 32; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }
    
    /* 第三步：warp内规约
    - 最后32个元素的规约由一个warp完成
    - 使用专门的warp级优化函数
    - 避免了多余的同步操作
    */
    if (tid < 32) {
        WarpSharedMemReduce<T, blockSize>(smem, tid);
    }
    
    /* 第四步：存储结果
    - 每个块的最终结果存储在smem[0]
    - 由线程0写回全局内存
    */
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}




// 模板化的CPU规约函数
template<typename T>
void reduce_sum_cpu_kahan(T *x, T *out, int N) {
    T sum = T(0);
    T c = T(0);
    
    for (int i = 0; i < N; i++) {
        T y = x[i] - c;
        T t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }
    
    out[0] = sum;
}

// 模板化的最终规约函数
template<typename T>
T final_reduction_kahan(T *partial_sums, int size) {
    T sum = T(0);
    T c = T(0);
    
    for (int i = 0; i < size; i++) {
        T y = partial_sums[i] - c;
        T t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }
    
    return sum;
}

// 模板化的数据初始化函数
template<typename T>
void initialize_data(T *x, int N) {
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        x[i] = static_cast<T>(rand() % 1000) / static_cast<T>(100);
    }
}

// 模板化的结果验证函数
template<typename T>
bool verify_results(T *gpu_result, T *cpu_result, int N) {
    T relative_error = std::abs(cpu_result[0] - gpu_result[0]) / cpu_result[0];
    if (relative_error > std::numeric_limits<T>::epsilon() * 100) {
        printf("Result verification failed!\n");
        printf("Relative Error: %e\n", static_cast<double>(relative_error));
        return false;
    }
    return true;
}

// 模板化的资源清理函数
template<typename T>
void cleanup(T *d_in, T *d_out, T *h_in, T *h_out, T *h_cpu_out) {
    if (d_in) hipFree(d_in);
    if (d_out) hipFree(d_out);
    if (h_in) free(h_in);
    if (h_out) free(h_out);
    if (h_cpu_out) free(h_cpu_out);
}

// 模板化的主计算函数
template<typename T>
void run_reduction(int N) {
    const int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize / 2;
    size_t nbytes = N * sizeof(T);
    size_t out_nbytes = gridSize * sizeof(T);

    // 声明指针
    T *d_in = NULL, *d_out = NULL;
    T *h_in = NULL, *h_out = NULL, *h_cpu_out = NULL;

    // 分配内存
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_in, nbytes));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_out, out_nbytes));

    h_in = (T *)malloc(nbytes);
    h_out = (T *)malloc(out_nbytes);
    h_cpu_out = (T *)malloc(sizeof(T));
    
    if (!h_in || !h_out || !h_cpu_out) {
        printf("CPU Memory allocation failed!\n");
        cleanup(d_in, d_out, h_in, h_out, h_cpu_out);
        return;
    }

    // 初始化数据
    initialize_data(h_in, N);

    // 复制数据到GPU
    CHECK_CUDA_ERROR(hipMemcpy(d_in, h_in, nbytes, hipMemcpyHostToDevice));

    // GPU计时
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    
    float gpu_time = 0.0f;
    CHECK_CUDA_ERROR(hipEventRecord(start));
    
    // 调用kernel
    reduce_warpseme<T, blockSize><<<gridSize, blockSize>>>(d_in, d_out, N);
    
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&gpu_time, start, stop));
    
    // 检查kernel执行
    CHECK_CUDA_ERROR(hipGetLastError());
    
    // 复制结果回CPU
    CHECK_CUDA_ERROR(hipMemcpy(h_out, d_out, out_nbytes, hipMemcpyDeviceToHost));

    // CPU计算
    reduce_sum_cpu_kahan(h_in, h_cpu_out, N);

    // 最终规约
    T final_result = final_reduction_kahan(h_out, gridSize);
    h_out[0] = final_result;

    // 验证结果
    if (verify_results(h_out, h_cpu_out, 1)) {
        printf("Results verified successfully!\n");
        printf("GPU Execution time: %f ms\n", gpu_time);
        printf("Final sum: %e\n", static_cast<double>(h_out[0]));
    }
    else {
        printf("Results verification failed!\n");
        printf("CPU result: %e\n", static_cast<double>(h_cpu_out[0]));
        printf("GPU result: %e\n", static_cast<double>(h_out[0]));
    }

    // 清理资源
    cleanup(d_in, d_out, h_in, h_out, h_cpu_out);
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
}

int main() {
    const int N = 25600000;
    
    printf("\nRunning with float:\n");
    run_reduction<float>(N);
    
    printf("\nRunning with double:\n");
    run_reduction<double>(N);
    
    printf("\nRunning with int:\n");
    run_reduction<int>(N);
    
    return 0;
}